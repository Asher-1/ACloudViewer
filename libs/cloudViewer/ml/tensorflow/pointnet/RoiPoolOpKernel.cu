// ----------------------------------------------------------------------------
// -                        CloudViewer: www.cloudViewer.org                  -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.cloudViewer.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------

#define EIGEN_USE_GPU
#include "RoiPoolOpKernel.h"
#include "ml/Helper.h"
#include "ml/contrib/RoiPoolKernel.h"

using namespace cloudViewer;
using namespace cloudViewer::ml;
using namespace cloudViewer::ml::contrib;
using namespace tensorflow;

class RoiPoolOpKernelCUDA : public RoiPoolOpKernel {
public:
    explicit RoiPoolOpKernelCUDA(OpKernelConstruction *construction)
        : RoiPoolOpKernel(construction) {}

    void Kernel(tensorflow::OpKernelContext *context,
                int batch_size,
                int pts_num,
                int boxes_num,
                int feature_in_len,
                int sampled_pts_num,
                const float *xyz,
                const float *boxes3d,
                const float *pts_feature,
                float *pooled_features,
                int *pooled_empty_flag) {
        hipError_t err;

        hipMemset(pooled_features, 0,
                   batch_size * boxes_num * sampled_pts_num *
                           (3 + feature_in_len) * sizeof(float));
        hipMemset(pooled_empty_flag, 0, batch_size * boxes_num * sizeof(int));

        roipool3dLauncher(batch_size, pts_num, boxes_num, feature_in_len,
                          sampled_pts_num, xyz, boxes3d, pts_feature,
                          pooled_features, pooled_empty_flag);

        // hipDeviceSynchronize();  // for using printf in kernel function
        err = hipGetLastError();
        if (hipSuccess != err) {
            fprintf(stderr, "CUDA kernel failed : %s\n",
                    hipGetErrorString(err));
            exit(-1);
        }
    }
};

REGISTER_KERNEL_BUILDER(Name("CloudviewerRoiPool").Device(DEVICE_GPU),
                        RoiPoolOpKernelCUDA);
