// ----------------------------------------------------------------------------
// -                        CloudViewer: www.cloudViewer.org                  -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.cloudViewer.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------

#include "core/nns/kernel/BlockSelectImpl.cuh"

namespace cloudViewer {
namespace core {
BLOCK_SELECT_IMPL(double, int32_t, true, 1, 1);
BLOCK_SELECT_IMPL(double, int32_t, false, 1, 1);

BLOCK_SELECT_IMPL(double, int32_t, true, 32, 2);
BLOCK_SELECT_IMPL(double, int32_t, false, 32, 2);

BLOCK_SELECT_IMPL(double, int32_t, true, 64, 3);
BLOCK_SELECT_IMPL(double, int32_t, false, 64, 3);

BLOCK_SELECT_IMPL(double, int32_t, true, 128, 3);
BLOCK_SELECT_IMPL(double, int32_t, false, 128, 3);

BLOCK_SELECT_IMPL(double, int32_t, true, 256, 4);
BLOCK_SELECT_IMPL(double, int32_t, false, 256, 4);

BLOCK_SELECT_IMPL(double, int32_t, true, 512, 8);
BLOCK_SELECT_IMPL(double, int32_t, false, 512, 8);

BLOCK_SELECT_IMPL(double, int32_t, true, 1024, 8);
BLOCK_SELECT_IMPL(double, int32_t, false, 1024, 8);

#if GPU_MAX_SELECTION_K >= 2048
BLOCK_SELECT_IMPL(double, int32_t, true, 2048, 8);
BLOCK_SELECT_IMPL(double, int32_t, false, 2048, 8);
#endif

BLOCK_SELECT_IMPL(double, int64_t, true, 1, 1);
BLOCK_SELECT_IMPL(double, int64_t, false, 1, 1);

BLOCK_SELECT_IMPL(double, int64_t, true, 32, 2);
BLOCK_SELECT_IMPL(double, int64_t, false, 32, 2);

BLOCK_SELECT_IMPL(double, int64_t, true, 64, 3);
BLOCK_SELECT_IMPL(double, int64_t, false, 64, 3);

BLOCK_SELECT_IMPL(double, int64_t, true, 128, 3);
BLOCK_SELECT_IMPL(double, int64_t, false, 128, 3);

BLOCK_SELECT_IMPL(double, int64_t, true, 256, 4);
BLOCK_SELECT_IMPL(double, int64_t, false, 256, 4);

BLOCK_SELECT_IMPL(double, int64_t, true, 512, 8);
BLOCK_SELECT_IMPL(double, int64_t, false, 512, 8);

BLOCK_SELECT_IMPL(double, int64_t, true, 1024, 8);
BLOCK_SELECT_IMPL(double, int64_t, false, 1024, 8);

#if GPU_MAX_SELECTION_K >= 2048
BLOCK_SELECT_IMPL(double, int64_t, true, 2048, 8);
BLOCK_SELECT_IMPL(double, int64_t, false, 2048, 8);
#endif

void runBlockSelectPair(hipStream_t stream,
                        double* inK,
                        int32_t* inV,
                        double* outK,
                        int32_t* outV,
                        bool dir,
                        int k,
                        int dim,
                        int num_points) {
    CLOUDVIEWER_ASSERT(k <= GPU_MAX_SELECTION_K);

    if (dir) {
        if (k == 1) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, true, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, true, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, true, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, true, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, true, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, true, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, true, 2048);
#endif
        }
    } else {
        if (k == 1) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, false, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, false, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, false, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, false, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, false, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, false, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_PAIR_CALL(double, int32_t, false, 2048);
#endif
        }
    }
}

void runBlockSelectPair(hipStream_t stream,
                        double* inK,
                        int64_t* inV,
                        double* outK,
                        int64_t* outV,
                        bool dir,
                        int k,
                        int dim,
                        int num_points) {
    CLOUDVIEWER_ASSERT(k <= GPU_MAX_SELECTION_K);

    if (dir) {
        if (k == 1) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, true, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, true, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, true, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, true, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, true, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, true, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, true, 2048);
#endif
        }
    } else {
        if (k == 1) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, false, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, false, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, false, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, false, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, false, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, false, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_PAIR_CALL(double, int64_t, false, 2048);
#endif
        }
    }
}

}  // namespace core
}  // namespace cloudViewer
