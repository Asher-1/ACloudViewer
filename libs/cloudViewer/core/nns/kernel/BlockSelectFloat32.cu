// ----------------------------------------------------------------------------
// -                        CloudViewer: www.cloudViewer.org                  -
// ----------------------------------------------------------------------------
// Copyright (c) 2018-2024 www.cloudViewer.org
// SPDX-License-Identifier: MIT
// ----------------------------------------------------------------------------

#include "core/nns/kernel/BlockSelectImpl.cuh"

namespace cloudViewer {
namespace core {
BLOCK_SELECT_IMPL(float, int32_t, true, 1, 1);
BLOCK_SELECT_IMPL(float, int32_t, false, 1, 1);

BLOCK_SELECT_IMPL(float, int32_t, true, 32, 2);
BLOCK_SELECT_IMPL(float, int32_t, false, 32, 2);

BLOCK_SELECT_IMPL(float, int32_t, true, 64, 3);
BLOCK_SELECT_IMPL(float, int32_t, false, 64, 3);

BLOCK_SELECT_IMPL(float, int32_t, true, 128, 3);
BLOCK_SELECT_IMPL(float, int32_t, false, 128, 3);

BLOCK_SELECT_IMPL(float, int32_t, true, 256, 4);
BLOCK_SELECT_IMPL(float, int32_t, false, 256, 4);

BLOCK_SELECT_IMPL(float, int32_t, true, 512, 8);
BLOCK_SELECT_IMPL(float, int32_t, false, 512, 8);

BLOCK_SELECT_IMPL(float, int32_t, true, 1024, 8);
BLOCK_SELECT_IMPL(float, int32_t, false, 1024, 8);

#if GPU_MAX_SELECTION_K >= 2048
BLOCK_SELECT_IMPL(float, int32_t, true, 2048, 8);
BLOCK_SELECT_IMPL(float, int32_t, false, 2048, 8);
#endif

BLOCK_SELECT_IMPL(float, int64_t, true, 1, 1);
BLOCK_SELECT_IMPL(float, int64_t, false, 1, 1);

BLOCK_SELECT_IMPL(float, int64_t, true, 32, 2);
BLOCK_SELECT_IMPL(float, int64_t, false, 32, 2);

BLOCK_SELECT_IMPL(float, int64_t, true, 64, 3);
BLOCK_SELECT_IMPL(float, int64_t, false, 64, 3);

BLOCK_SELECT_IMPL(float, int64_t, true, 128, 3);
BLOCK_SELECT_IMPL(float, int64_t, false, 128, 3);

BLOCK_SELECT_IMPL(float, int64_t, true, 256, 4);
BLOCK_SELECT_IMPL(float, int64_t, false, 256, 4);

BLOCK_SELECT_IMPL(float, int64_t, true, 512, 8);
BLOCK_SELECT_IMPL(float, int64_t, false, 512, 8);

BLOCK_SELECT_IMPL(float, int64_t, true, 1024, 8);
BLOCK_SELECT_IMPL(float, int64_t, false, 1024, 8);

#if GPU_MAX_SELECTION_K >= 2048
BLOCK_SELECT_IMPL(float, int64_t, true, 2048, 8);
BLOCK_SELECT_IMPL(float, int64_t, false, 2048, 8);
#endif

void runBlockSelectPair(hipStream_t stream,
                        float* inK,
                        int32_t* inV,
                        float* outK,
                        int32_t* outV,
                        bool dir,
                        int k,
                        int dim,
                        int num_points) {
    CLOUDVIEWER_ASSERT(k <= GPU_MAX_SELECTION_K);

    if (dir) {
        if (k == 1) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, true, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, true, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, true, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, true, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, true, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, true, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, true, 2048);
#endif
        }
    } else {
        if (k == 1) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, false, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, false, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, false, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, false, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, false, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, false, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_PAIR_CALL(float, int32_t, false, 2048);
#endif
        }
    }
}

void runBlockSelectPair(hipStream_t stream,
                        float* inK,
                        int64_t* inV,
                        float* outK,
                        int64_t* outV,
                        bool dir,
                        int k,
                        int dim,
                        int num_points) {
    CLOUDVIEWER_ASSERT(k <= GPU_MAX_SELECTION_K);

    if (dir) {
        if (k == 1) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, true, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, true, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, true, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, true, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, true, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, true, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, true, 2048);
#endif
        }
    } else {
        if (k == 1) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, false, 1);
        } else if (k <= 32) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, false, 32);
        } else if (k <= 64) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, false, 64);
        } else if (k <= 128) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, false, 128);
        } else if (k <= 256) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, false, 256);
        } else if (k <= 512) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, false, 512);
        } else if (k <= 1024) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            BLOCK_SELECT_PAIR_CALL(float, int64_t, false, 2048);
#endif
        }
    }
}

}  // namespace core
}  // namespace cloudViewer
